#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdint.h>
#include <sys/time.h>
#include "boruvka_mst_gpu.h"

// Some useful macros
#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a < b ? b : a)
#define CEILING_DIV(a, b) ((a + b - 1) / b)
#define MAX_INT ((int) 0x7fffffff)

// parallel memcpy for device context
/*
__global__ static void myMemcpyHelper(int * dst, int *src) {
    int i = threadIdx.x;
    dst[i] = src[i];
}

__device__ static void myMemcpy(int *dst, int *src, int length) {
    myMemcpyHelper<<<1, length>>>(dst, src);
    hipDeviceSynchronize();
}
*/

// helper function for findMin()
__global__ static void minReduce(int *src, int *dest, int len) {
    int i = threadIdx.x;
    if (2 * i + 1 < len) {
        dest[i] = MIN(src[2 * i], src[2 * i + 1]);
        //printf("at i = %d, min of %d and %d is %d\n", i, src[2 * i], src[2 * i + 1], dest[i]);
    } else {
        dest[i] = src[2 * i];
    }
}

// function to find the min of an array using reduce
__device__ static void findMin(int *arr, int len, int *min) {
    //printf("len = %d\n", len);
    if (len == 1) {
        //printf("found min: %d\n", arr[0]);
        *min = arr[0];
    } else if (len > 1) {
        int *arr2;
        int newLen = CEILING_DIV(len, 2);
        hipMalloc((void **)&arr2, newLen * sizeof(int));
        minReduce<<<1, newLen>>>(arr, arr2, len);
        hipDeviceSynchronize();
        findMin(arr2, newLen, min);
        hipFree(arr2);
    } 
}

/*
__device__ static void findMin(int *arr, int len, int *min) {
    int *src;
    int *dest;
    hipMalloc(&src, len * sizeof(int));
    myMemcpy(src, arr, len);
    hipMalloc(&dest, len * sizeof(int));
    for (int size = len; size > 1; size = CEILING_DIV(size, 2)) {
        for (int i = 0; i < CEILING_DIV(size, 2); ++i) {
            if (2 * i + 1 < size) {
                dest[i] = MIN(src[2 * i], src[2 * i + 1]);
                printf("at i = %d, min of %d and %d is %d\n", i, src[2 * i], src[2 * i + 1], dest[i]);
            } else {
                dest[i] = src[2 * i];
            }
        }
        myMemcpy(src, dest, CEILING_DIV(size, 2));
        printf("size: %d\n", size);
    }
    *min = MIN(dest[0], dest[1]);
    hipFree(src);
    hipFree(dest);
}
*/

__device__ static void findMinSeq(int *arr, int len, int *min) {
    int n = MAX_INT;
    for (int i = 0; i < len; ++i) {
        if (arr[i] < n) {
            n = arr[i];
        }
    }
    *min = n;
}

// function to find the index a specified value appears in an array
__global__ static void findIdx(int *arr, int val, int *idx) {
    int i = threadIdx.x;
    //printf("Searching for %d\n", val);
    if (arr[i] == val) {
        *idx = i;
    }
}

// function to create pseudo-trees (first for loop in handout)
__global__ static void getPseudoTree(int *graph, int *T, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    if (exists[v]) {
        // find w such that (v, w) is the minimum weight edge of v.
        int *minEdgeWeight;
        int *minEdgeVertex;

        hipMalloc(&minEdgeWeight, sizeof(int));
        hipMalloc(&minEdgeVertex, sizeof(int));

        int *adjacencyList = graph + v * numVertices;

        // solve weird bug with self-loops
        adjacencyList[v] = MAX_INT;

        // for debugging
        *minEdgeWeight = adjacencyList[0];
        *minEdgeVertex = 0;
        
        findMin(adjacencyList, numVertices, minEdgeWeight);
        findIdx<<<1,numVertices>>>(adjacencyList, *minEdgeWeight, minEdgeVertex);
        hipDeviceSynchronize();

        //printf("parent of %d is %d with weight %d\n", v, *minEdgeVertex, *minEdgeWeight);

        // update the parent
        parent[v] = *minEdgeVertex;

        // Update the minimum spanning tree. Since there are two copies of each edge in the matrix, only update the earlier one
        T[MIN(v, *minEdgeVertex) * numVertices + MAX(v, *minEdgeVertex)] = *minEdgeWeight;    // T := T U {(v, w)}
        //printf("adding edge with weight %d between %d and %d\n", *minEdgeWeight, v, *minEdgeVertex);

        // simple version
        //T[v * numVertices + minEdgeVertex] = minEdgeWeight;
        //T[minEdgeVertex * numVertices + v] = minEdgeWeight;
    }
}

__global__ static void getPseudoTreeSeq(int *graph, int *T, int *parent, int numVertices, bool *exists) {
    for (int v = 0; v < numVertices; ++v) {
        if (exists[v]) {
            // find w such that (v, w) is the minimum weight edge of v.
            int *minEdgeWeight;
            int *minEdgeVertex;

            hipMalloc(&minEdgeWeight, sizeof(int));
            hipMalloc(&minEdgeVertex, sizeof(int));

            int *adjacencyList = graph + v * numVertices;

            // for debugging
            *minEdgeWeight = adjacencyList[0];
            *minEdgeVertex = 0;
            
            findMinSeq(adjacencyList, numVertices, minEdgeWeight);
            for (int i = 0; i < numVertices; ++i) {
                if (adjacencyList[i] == *minEdgeWeight) {
                    *minEdgeVertex = i;
                }
            }

            // update the parent
            parent[v] = *minEdgeVertex;

            // Update the minimum spanning tree. Since there are two copies of each edge in the matrix, only update the earlier one
            T[MIN(v, *minEdgeVertex) * numVertices + MAX(v, *minEdgeVertex)] = *minEdgeWeight;    // T := T U {(v, w)}
        }
    }
}

// function to convert pseudo trees into rooted trees (second for loop in handout)
__global__ static void makeRootedTrees(int *parent, bool *exists) {
    int v = threadIdx.x;
    //printf("making rooted tree at %d\n", v);
    if ((exists[v]) && (parent[parent[v]] == v) && (v < parent[v])) {
        //printf("updating parent for %d\n", v);
        parent[v] = v;
    }
}

__global__ static void makeRootedTreesSeq(int *parent, bool *exists, int numVertices) {
    int v = threadIdx.x;
    //printf("making rooted tree at %d\n", v);
    for (int v = 0; v < numVertices; ++v) {
        if ((exists[v]) && (parent[parent[v]] == v) && (v < parent[v])) {
            //printf("updating parent for %d\n", v);
            parent[v] = v;
        }
    }
}

// function to convert rooted trees into rooted stars (third for loop in handout)
__global__ static void makeRootedStars(int *parent, bool *exists) {
    int v = threadIdx.x;

    if (exists[v]) {
        while (parent[v] != parent[parent[v]]) {
            parent[v] = parent[parent[v]];
        }
    }
}

__global__ static void makeRootedStarsSeq(int *parent, bool *exists, int numVertices) {
    for (int v = 0; v < numVertices; ++v) {
        if (exists[v]) {
            while (parent[v] != parent[parent[v]]) {
                parent[v] = parent[parent[v]];
            }
        }
    }
}

// helper function for contractRootedStars() to remove all edges connected to a specified vertex
__global__ static void removeEdges(int *graph, int v, int numVertices) {
    int u = threadIdx.x;

    graph[u * numVertices + v] = MAX_INT;
    graph[v * numVertices + u] = MAX_INT;
}

__global__ static void transferVertexEdgesToParent(int *graph, int numVertices, int *parent, int v) {
    int i = threadIdx.x;
    
    if ((graph[v * numVertices + i] != MAX_INT) && (i != parent[v])) {
        // while loop instead of if statement is a hack to fix race condition
        while ((graph[parent[v] * numVertices + parent[i]] > graph[v * numVertices + i])    
                || (graph[parent[i] * numVertices + parent[v]] > graph[i * numVertices + v])) {
            graph[parent[v] * numVertices + parent[i]] = graph[v * numVertices + i];
            graph[parent[i] * numVertices + parent[v]] = graph[i * numVertices + v];
        }
    }
}

__global__ static void transferEdgesToParent(int *graph, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    // remove vertex if it is not the root of a rooted star
    if ((exists[v]) && (parent[v] != v)) {
        //printf("transferring edges from vertex %d\n", v);
        // TODO: parallelize this
        /*
        for (int i = 0; i < numVertices; ++i) {
            if ((graph[v * numVertices + i] != MAX_INT) && (i != parent[v])) {
                if (graph[parent[v] * numVertices + parent[i]] > graph[v * numVertices + i]) {
                    graph[parent[v] * numVertices + parent[i]] = graph[v * numVertices + i];
                    graph[parent[i] * numVertices + parent[v]] = graph[i * numVertices + v];
                }
            }
        }
        */

        transferVertexEdgesToParent<<<1,numVertices>>>(graph, numVertices, parent, v);
        hipDeviceSynchronize();
    }
}

/*
// do the same thing as transferEdgesToParent but without calling transferVertexEdgesToParent
__global__ static void transferAllEdgesToParent(int *graph, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x / numVertices;
    int i = threadIdx.x % numVertices;

    if ((exists[v]) && (parent[v] != v)) {
        if ((graph[v * numVertices + i] != MAX_INT) && (i != parent[v])) {
            // while loop instead of if statement is a hack to fix race condition
            while ((graph[parent[v] * numVertices + parent[i]] > graph[v * numVertices + i])    
                    || (graph[parent[i] * numVertices + parent[v]] > graph[i * numVertices + v])) {
                graph[parent[v] * numVertices + parent[i]] = graph[v * numVertices + i];
                graph[parent[i] * numVertices + parent[v]] = graph[i * numVertices + v];
            }
            hipDeviceSynchronize();
        }
    }
}
*/

__global__ static void transferEdgesSeq(int *graph, int *parent, int numVertices, bool *exists) {
    for (int v = 0; v < numVertices; ++v) {
        for (int i = 0; i < numVertices; ++i) {
            if ((exists[v]) && (parent[v] != v) && (graph[v * numVertices + i] != MAX_INT) && (i != parent[v])) {
                if ((graph[parent[v] * numVertices + parent[i]] > graph[v * numVertices + i])    
                    || (graph[parent[i] * numVertices + parent[v]] > graph[i * numVertices + v])) {
                        graph[parent[v] * numVertices + parent[i]] = graph[v * numVertices + i];
                        graph[parent[i] * numVertices + parent[v]] = graph[i * numVertices + v];
                    }
            }
        }
    }
}

// function to contract all rooted stars
__global__ static void contractRootedStars(int *graph, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    // remove vertex if it is not the root of a rooted star
    if ((exists[v]) && (parent[v] != v)) {
        //printf("contracting vertex %d\n", v);
        exists[v] = false;
        // remove edges that connect to this vertex
        removeEdges<<<1,numVertices>>>(graph, v, numVertices);
        //hipDeviceSynchronize();
        /*
        for (int i = 0; i < numVertices; ++i) {
            graph[v * numVertices + i] = MAX_INT;
            graph[i * numVertices + v] = MAX_INT;
        }
        */
    }
}

__global__ static void contractRootedStarsSeq(int *graph, int *parent, int numVertices, bool *exists) {
    for (int v = 0; v < numVertices; ++v) {
        if ((exists[v]) && (parent[v] != v)) {
            //printf("contracting vertex %d\n", v);
            exists[v] = false;
            // remove edges that connect to this vertex
            for (int i = 0; i < numVertices; ++i) {
                graph[v * numVertices + i] = MAX_INT;
                graph[i * numVertices + v] = MAX_INT;
            }
        }
    }
}

// helper function for sum()
template<typename T>
__global__ void sumReduce(T *src, T *dst, int len) {
    int i = threadIdx.x;
    if (2 * i + 1 < len) {
        dst[i] = src[2 * i] + src[2 * i + 1];
    } else {
        dst[i] = src[2 * i];
    }
}

// function to find the sum of an array using reduce
template<typename T>
__global__ void sum(T *arr, int len, int *result) {
    if (len > 1) {
        T *arr2;
        int newLen = CEILING_DIV(len, 2);
        hipMalloc((void **)&arr2, newLen * sizeof(T));
        sumReduce<T><<<1, newLen>>>(arr, arr2, len);
        hipDeviceSynchronize();
        sum<T><<<1,1>>>(arr2, newLen, result);
        hipFree(arr2);
    } else {
        *result = arr[0];
    }
}


__global__ void sumByte(uint8_t *arr, int len, int *result) {
    int n = 0;
    for (int i = 0; i < len; ++i) {
        n += arr[i];
        //printf("found %d\n", arr[i]);
    }
    //printf("sum: %d\n", n);
    *result = n;
}

__global__ void sumInt(int *arr, int len, int *result) {
    int n = 0;
    for (int i = 0; i < len; ++i) {
        n += arr[i];
    }
    *result = n;
}

__device__ void printTreeHelper(int *T, int numVertices) {
    for (int v = 0; v < numVertices; ++v) {
        for (int e = 0; e < numVertices; ++e) {
            if (T[v * numVertices + e] != 0) {
                printf("MST edge has weight of size %d between %d and %d\n", T[v * numVertices + e], v, e);
            }
        }
    }
}

__global__ void printTree(int *T, int numVertices) {
    printTreeHelper(T, numVertices);
}

// main function for Boruvka's algorithm
int boruvka(Graph &g, int &time) {
    // copy graph to GPU
    int *graph;
    int numVertices = g.size();
    size_t graphSize = numVertices * numVertices * sizeof(int);
    hipMalloc((void **) &graph, graphSize);
    hipMemcpy(graph, g.raw(), graphSize, hipMemcpyHostToDevice);

    //printf("copied graph to GPU\n");

    // start timer
    // from https://stackoverflow.com/questions/1952290/how-can-i-get-utctime-in-millisecond-since-january-1-1970-in-c-language
    struct timeval tv;
    gettimeofday(&tv, NULL);
    unsigned long long startTime =
    (unsigned long long)(tv.tv_sec) * 1000 +
    (unsigned long long)(tv.tv_usec) / 1000;

    // set up array to hold mst
    int *T;
    hipMalloc((void **) &T, graphSize);
    hipMemset((void *) T, 0, graphSize);

    // set up parent array
    int *parent;
    hipMalloc((void **) &parent, g.size() * sizeof(int));
    hipMemset((void *) parent, 0, g.size() * sizeof(int));

    // set up device vector to mark vertices as existant
    bool *exists;
    hipMalloc((void **) &exists, g.size() * sizeof(bool));
    hipMemset((void *) exists, true, g.size() * sizeof(bool));

    //printf("Initialized global arrays\n");

    int numExistingVertices = numVertices;
    int *d_numExistingVertices;
    hipMalloc((void **)&d_numExistingVertices, sizeof(int));
    while (numExistingVertices > 1) {
        //printf("%d vertices remaining\n", numExistingVertices);
        // get pseudo-tree from the graph
        getPseudoTree<<<1,numVertices>>>(graph, T, parent, numVertices, exists);
        hipDeviceSynchronize();
        //getPseudoTreeSeq<<<1,1>>>(graph, T, parent, numVertices, exists);

        // convert pseudo-trees to rooted trees
        makeRootedTrees<<<1,numVertices>>>(parent, exists);
        //makeRootedTreesSeq<<<1,1>>>(parent, exists, numVertices);
        hipDeviceSynchronize();

        // convert every rooted tree into a rooted star
        makeRootedStars<<<1,numVertices>>>(parent, exists);
        //makeRootedStarsSeq<<<1,1>>>(parent, exists, numVertices);
        hipDeviceSynchronize();

        // contract all rooted stars into a single vertex
        transferEdgesToParent<<<1,numVertices>>>(graph, parent, numVertices, exists);
        hipDeviceSynchronize();
        //transferAllEdgesToParent<<<1,numVertices * numVertices>>>(graph, parent, numVertices, exists);
        //transferEdgesSeq<<<1,1>>>(graph, parent, numVertices, exists);
        contractRootedStars<<<1,numVertices>>>(graph, parent, numVertices, exists);
        //contractRootedStarsSeq<<<1,1>>>(graph, parent, numVertices, exists);
        hipDeviceSynchronize();

        // update number of existing vertices
        hipMemcpy(d_numExistingVertices, &numExistingVertices, sizeof(int), hipMemcpyHostToDevice);
        sumByte<<<1,1>>>((uint8_t *)exists, numVertices, d_numExistingVertices);
        hipMemcpy(&numExistingVertices, d_numExistingVertices, sizeof(int), hipMemcpyDeviceToHost);
    }

    //printTree<<<1,1>>>(T, numVertices);

    // return the total weight of the minimum spanning tree
    int result;
    int *d_result;
    hipMalloc((void **)&d_result, sizeof(int));
    sumInt<<<1,1>>>(T, numVertices * numVertices, d_result);
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // clean up
    hipFree(graph);
    hipFree(T);
    hipFree(parent);
    hipFree(exists);
    hipFree(d_numExistingVertices);
    hipFree(d_result);

    // from https://stackoverflow.com/questions/1952290/how-can-i-get-utctime-in-millisecond-since-january-1-1970-in-c-language
    gettimeofday(&tv, NULL);
    unsigned long long endTime =
    (unsigned long long)(tv.tv_sec) * 1000 +
    (unsigned long long)(tv.tv_usec) / 1000;
    time = endTime - startTime;

    return result;
}
