#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdint.h>
#include "boruvka_mst_gpu.h"

// Some useful macros
#define MIN(a, b) (a < b ? a : b)
#define MAX(a, b) (a < b ? b : a)
#define CEILING_DIV(a, b) ((a + b - 1) / b)
#define MAX_INT ((int) 0x7fffffff)

// helper function for findMin()
__global__ static void minReduce(int *src, int *dest, int len) {
    int i = threadIdx.x;
    if (2 * i + 1 < len) {
        dest[i] = MIN(src[2 * i], src[2 * i + 1]);
    } else {
        dest[i] = src[2 * i];
    }
}

// function to find the min of an array using reduce
__device__ static void findMin(int *arr, int len, int *min) {
    if (len > 1) {
        int *arr2;
        int newLen = CEILING_DIV(len, 2);
        hipMalloc((void **)&arr2, newLen * sizeof(int));
        minReduce<<<1, newLen>>>(arr, arr2, len);
        findMin(arr2, newLen, min);
        hipFree(arr2);
    } else {
        *min = arr[0];
    }
}

/*
__device__ static void findMin(int *arr, int len, int *min) {
    int n = MAX_INT;
    for (int i = 0; i < len; ++i) {
        if (arr[i] < n) {
            n = arr[i];
        }
    }
    *min = n;
}
*/

// function to find the index a specified value appears in an array
__global__ static void findIdx(int *arr, int val, int *idx) {
    int i = threadIdx.x;
    printf("Searching for %d\n", val);
    if (arr[i] == val) {
        *idx = i;
    }
}

// function to create pseudo-trees (first for loop in handout)
__global__ static void getPseudoTree(int *graph, int *T, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    if (exists[v]) {
        // find w such that (v, w) is the minimum weight edge of v.
        int minEdgeWeight;
        int minEdgeVertex;

        int *adjacencyList = graph + v * numVertices;

        // for debugging
        minEdgeWeight = adjacencyList[0];
        minEdgeVertex = 0;

        for (int i = 1; i < numVertices; ++i) {
            if (adjacencyList[i] < minEdgeWeight) {
                minEdgeVertex = i;
                minEdgeWeight = adjacencyList[i];
            }
        }
        
        /*
        findMin(&(graph[v * numVertices]), numVertices, &minEdgeWeight);
        findIdx<<<1,numVertices>>>(adjacencyList, minEdgeWeight, &minEdgeVertex);
        */

        printf("parent of %d is %d with weight %d\n", v, minEdgeVertex, minEdgeWeight);

        // update the parent
        parent[v] = minEdgeVertex;

        // Update the minimum spanning tree. Since there are two copies of each edge in the matrix, only update the earlier one
        T[MIN(v, minEdgeVertex) * numVertices + MAX(v, minEdgeVertex)] = minEdgeWeight;    // T := T U {(v, w)}

        // simple version
        //T[v * numVertices + minEdgeVertex] = minEdgeWeight;
        //T[minEdgeVertex * numVertices + v] = minEdgeWeight;
    }
}

// function to convert pseudo trees into rooted trees (second for loop in handout)
__global__ static void makeRootedTrees(int *parent, bool *exists) {
    int v = threadIdx.x;
    //printf("making rooted tree at %d\n", v);
    if ((exists[v]) && (parent[parent[v]] == v) && (v < parent[v])) {
        printf("updating parent for %d\n", v);
        parent[v] = v;
    }
}

// function to convert rooted trees into rooted stars (third for loop in handout)
__global__ static void makeRootedStars(int *parent, bool *exists) {
    int v = threadIdx.x;

    if (exists[v]) {
        while (parent[v] != parent[parent[v]]) {
            parent[v] = parent[parent[v]];
        }
    }
}

// helper function for contractRootedStars() to remove all edges connected to a specified vertex
__global__ static void removeEdges(int *graph, int v, int numVertices) {
    int u = threadIdx.x;

    graph[u * numVertices + v] = MAX_INT;
    graph[v * numVertices + u] = MAX_INT;
}

__global__ static void transferEdgesToParent(int *graph, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    // remove vertex if it is not the root of a rooted star
    if ((exists[v]) && (parent[v] != v)) {
        printf("transferring edges from vertex %d\n", v);
        // TODO: parallelize this
        for (int i = 0; i < numVertices; ++i) {
            if ((graph[v * numVertices + i] != MAX_INT) && (i != parent[v])) {
                if (graph[parent[v] * numVertices + parent[i]] > graph[v * numVertices + i]) {
                    graph[parent[v] * numVertices + parent[i]] = graph[v * numVertices + i];
                    graph[parent[i] * numVertices + parent[v]] = graph[i * numVertices + v];
                }
            }
        }
    }
}

// function to contract all rooted stars
__global__ static void contractRootedStars(int *graph, int *parent, int numVertices, bool *exists) {
    int v = threadIdx.x;

    // remove vertex if it is not the root of a rooted star
    if ((exists[v]) && (parent[v] != v)) {
        printf("contracting vertex %d\n", v);
        exists[v] = false;
        // remove edges that connect to this vertex
        //removeEdges<<<1,numVertices>>>(graph, v, numVertices);
        for (int i = 0; i < numVertices; ++i) {
            graph[v * numVertices + i] = MAX_INT;
            graph[i * numVertices + v] = MAX_INT;
        }
    }
}

// helper function for sum()
template<typename T>
__global__ void sumReduce(T *src, T *dst, int len) {
    int i = threadIdx.x;
    if (2 * i + 1 < len) {
        dst[i] = src[2 * i] + src[2 * i + 1];
    } else {
        dst[i] = src[2 * i];
    }
}

// function to find the sum of an array using reduce
/*
template<typename T>
__global__ void sum(T *arr, int len, int *result) {
    if (len > 1) {
        T *arr2;
        int newLen = CEILING_DIV(len, 2);
        hipMalloc((void **)&arr2, newLen * sizeof(T));
        sumReduce<T><<<1, newLen>>>(arr, arr2, len);
        sum<T><<<1,1>>>(arr2, newLen, result);
        hipFree(arr2);
    } else {
        *result = arr[0];
    }
}
*/


__global__ void sumByte(uint8_t *arr, int len, int *result) {
    int n = 0;
    for (int i = 0; i < len; ++i) {
        n += arr[i];
        //printf("found %d\n", arr[i]);
    }
    //printf("sum: %d\n", n);
    *result = n;
}

__global__ void sumInt(int *arr, int len, int *result) {
    int n = 0;
    for (int i = 0; i < len; ++i) {
        n += arr[i];
    }
    *result = n;
}

// main function for Boruvka's algorithm
int boruvka(Graph &g) {
    // copy graph to GPU
    int *graph;
    int numVertices = g.size();
    size_t graphSize = numVertices * numVertices * sizeof(int);
    hipMalloc((void **) &graph, graphSize);
    hipMemcpy(graph, g.raw(), graphSize, hipMemcpyHostToDevice);

    printf("copied graph to GPU\n");

    // set up array to hold mst
    int *T;
    hipMalloc((void **) &T, graphSize);
    hipMemset((void *) T, 0, graphSize);

    // set up parent array
    int *parent;
    hipMalloc((void **) &parent, g.size() * sizeof(int));
    hipMemset((void *) parent, 0, g.size() * sizeof(int));

    // set up device vector to mark vertices as existant
    bool *exists;
    hipMalloc((void **) &exists, g.size() * sizeof(bool));
    hipMemset((void *) exists, true, g.size() * sizeof(bool));

    printf("Initialized global arrays\n");

    // for debugging
    void *buffer = malloc(graphSize);

    int numExistingVertices = numVertices;
    int *d_numExistingVertices;
    hipMalloc((void **)&d_numExistingVertices, sizeof(int));
    while (numExistingVertices > 1) {
        printf("%d vertices remaining\n", numExistingVertices);
        // get pseudo-tree from the graph
        getPseudoTree<<<1,numVertices>>>(graph, T, parent, numVertices, exists);

        // convert pseudo-trees to rooted trees
        makeRootedTrees<<<1,numVertices>>>(parent, exists);

        // convert every rooted tree into a rooted star
        makeRootedStars<<<1,numVertices>>>(parent, exists);

        // contract all rooted stars into a single vertex
        transferEdgesToParent<<<1,numVertices>>>(graph, parent, numVertices, exists);
        contractRootedStars<<<1,numVertices>>>(graph, parent, numVertices, exists);

        // update number of existing vertices
        hipMemcpy(d_numExistingVertices, &numExistingVertices, sizeof(int), hipMemcpyHostToDevice);
        sumByte<<<1,1>>>((uint8_t *)exists, numVertices, d_numExistingVertices);
        hipMemcpy(&numExistingVertices, d_numExistingVertices, sizeof(int), hipMemcpyDeviceToHost);
    }

    // return the total weight of the minimum spanning tree
    int result;
    int *d_result;
    hipMalloc((void **)&d_result, sizeof(int));
    sumInt<<<1,1>>>(T, numVertices * numVertices, d_result);
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // clean up
    hipFree(graph);
    hipFree(T);
    hipFree(parent);
    hipFree(exists);
    hipFree(d_numExistingVertices);
    hipFree(d_result);

    return result;
}
